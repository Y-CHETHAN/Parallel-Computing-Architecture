#include "hip/hip_runtime.h"
﻿#include <hip/hip_runtime.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <windows.h>
#include <>

#define N 1024
#define BLOCK_SIZE 16

__global__ void reduceUnrolling8(int* input, int* output)
{
    __shared__ int sharedData[BLOCK_SIZE];

    int tid = threadIdx.x;
    int i = blockIdx.x * blockDim.x * 8 + threadIdx.x;

    // Load data into shared memory
    sharedData[tid] = input[i] + input[i + blockDim.x] +
        input[i + 2 * blockDim.x] + input[i + 3 * blockDim.x] +
        input[i + 4 * blockDim.x] + input[i + 5 * blockDim.x] +
        input[i + 6 * blockDim.x] + input[i + 7 * blockDim.x];

    __syncthreads();

    // Perform reduction in shared memory
    for (unsigned int stride = blockDim.x / 2; stride > 0; stride >>= 1)
    {
        if (tid < stride)
        {
            sharedData[tid] += sharedData[tid + stride];
        }
        __syncthreads();
    }

    // Write the result back to global memory
    if (tid == 0)
    {
        output[blockIdx.x] = sharedData[0];
    }
}

__global__ void reduceUnrolling16(int* input, int* output)
{
    __shared__ int sharedData[BLOCK_SIZE];

    int tid = threadIdx.x;
    int i = blockIdx.x * blockDim.x * 16 + threadIdx.x;

    // Load data into shared memory
    sharedData[tid] = input[i] + input[i + blockDim.x] +
        input[i + 2 * blockDim.x] + input[i + 3 * blockDim.x] +
        input[i + 4 * blockDim.x] + input[i + 5 * blockDim.x] +
        input[i + 6 * blockDim.x] + input[i + 7 * blockDim.x] +
        input[i + 8 * blockDim.x] + input[i + 9 * blockDim.x] +
        input[i + 10 * blockDim.x] + input[i + 11 * blockDim.x] +
        input[i + 12 * blockDim.x] + input[i + 13 * blockDim.x] +
        input[i + 14 * blockDim.x] + input[i + 15 * blockDim.x];

    __syncthreads();

    // Perform reduction in shared memory
    for (unsigned int stride = blockDim.x / 2; stride > 0; stride >>= 1)
    {
        if (tid < stride)
        {
            sharedData[tid] += sharedData[tid + stride];
        }
        __syncthreads();
    }

    // Write the result back to global memory
    if (tid == 0)
    {
        output[blockIdx.x] = sharedData[0];
    }
}

int main()
{
    int input[N];
    int output;

    // Initialize input data
    for (int i = 0; i < N; i++)
    {
        input[i] = i;
    }

    int* d_input;
    int* d_output;
    hipMalloc((void**)&d_input, N * sizeof(int));
    hipMalloc((void**)&d_output, sizeof(int));

    // Copy input data to device
    hipMemcpy(d_input, input, N * sizeof(int), hipMemcpyHostToDevice);

    // Launch reduceUnrolling8 kernel and measure execution time with nvprof
    hipEvent_t start8, stop8;
    hipEventCreate(&start8);
    hipEventCreate(&stop8);

    hipEventRecord(start8);
    reduceUnrolling8 << <N / BLOCK_SIZE, BLOCK_SIZE >> > (d_input, d_output);
    hipEventRecord(stop8);
    hipEventSynchronize(stop8);

    // Copy result from device to host
    hipMemcpy(&output, d_output, sizeof(int), hipMemcpyDeviceToHost);
    printf("reduceUnrolling8 result: %d\n", output);

    float milliseconds8 = 0;
    hipEventElapsedTime(&milliseconds8, start8, stop8);
    printf("reduceUnrolling8 execution time: %.3f ms\n", milliseconds8);

    // Launch reduceUnrolling16 kernel and measure execution time with nvprof
    hipEvent_t start16, stop16;
    hipEventCreate(&start16);
    hipEventCreate(&stop16);

    hipEventRecord(start16);
    reduceUnrolling16 << <N / (BLOCK_SIZE * 16), BLOCK_SIZE >> > (d_input, d_output);
    hipEventRecord(stop16);
    hipEventSynchronize(stop16);

    // Copy result from device to host
    hipMemcpy(&output, d_output, sizeof(int), hipMemcpyDeviceToHost);
    printf("reduceUnrolling16 result: %d\n", output);

    float milliseconds16 = 0;
    hipEventElapsedTime(&milliseconds16, start16, stop16);
    printf("reduceUnrolling16 execution time: %.3f ms\n", milliseconds16);

    // Determine the kernel with the least execution time
    if (milliseconds8 < milliseconds16)
    {
        printf("reduceUnrolling8 has the least execution time.\n");
    }
    else if (milliseconds16 < milliseconds8)
    {
        printf("reduceUnrolling16 has the least execution time.\n");
    }
    else
    {
        printf("reduceUnrolling8 and reduceUnrolling16 have the same execution time.\n");
    }

    // Clean up
    hipFree(d_input);
    hipFree(d_output);

    return 0;
}
